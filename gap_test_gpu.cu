#include "hip/hip_runtime.h"
// Copyright 2021 Seth Troisi
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//      http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <algorithm>
#include <atomic>
#include <cassert>
#include <chrono>
#include <cmath>
#include <condition_variable>
#include <cstdint>
#include <cstdio>
#include <fstream>
#include <iostream>
#include <memory>
#include <mutex>
#include <sstream>
#include <string>
#include <thread>
#include <unistd.h>
#include <unordered_map>
#include <vector>

// pthread_setname_np
#include <pthread.h>

#include <gmp.h>

#include "gap_common.h"
#include "gap_test_common.h"
#include "miller_rabin.h"

using std::cout;
using std::endl;
using std::vector;
using namespace std::chrono;

#ifdef GPU_BITS
#define BITS GPU_BITS
#else
#define BITS 1024
#endif


#define WINDOW_BITS ((BITS <= 1024) ? 5 : 6)

/**
 * BATCH_GPU is 2^n >= 1024
 * SEQUENTIAL_IN_BATCH = {1,2,4}
 *      1 => 0 overhead
 *      2 => 0.5 extra PRP/m
 *      4 => 1.5 extra PRP/M
 *
 * BATCHED_M is number of M loaded at the same time
 * ------------------
 * Try:
 *  1024,   16384,  1,  8,  1
 *  ???2048,    4096,  2,  8,  1
 *  ???4096,    2048,  4,  16, 1
 *
 */
const size_t BATCH_GPU = 1024; //2*8192;
const size_t SEQUENTIAL_IN_BATCH = 2;
const size_t BATCHED_M = 2 * BATCH_GPU * 120 / 100 / SEQUENTIAL_IN_BATCH;  // 10% extra

/**
 * Originally 8 which has highest throughput but only if we have LOTS of instances
 * this helps reduce the number of parallel instances needed
 */
const int THREADS_PER_INSTANCE = 16;
const int ROUNDS = 1;

//************************************************************************

void prime_gap_test(const struct Config config);


int main(int argc, char* argv[]) {
    Config config = Args::argparse(argc, argv, Args::Pr::TEST_GPU);
    if (config.valid == 0) {
        Args::show_usage(argv[0], Args::Pr::TEST_GPU);
        return 1;
    }

    if (config.verbose >= 2) {
        printf("Compiled with GMP %d.%d.%d\n",
            __GNU_MP_VERSION, __GNU_MP_VERSION_MINOR, __GNU_MP_VERSION_PATCHLEVEL);
    }

    if( !has_prev_prime_gmp() ) {
        cout << "See Notes in README.md for instructions on using dev GMPlib" << endl;
        return 1;
    }

    if (config.sieve_length == 0) {
        cout << "Must set sieve-length for " << argv[0] << endl;
        Args::show_usage(argv[0], Args::Pr::TEST_GPU);
        return 1;
    }

    setlocale(LC_NUMERIC, "");
    if (config.verbose >= 0) {
        printf("\n");
        printf("Testing m * %d#/%d, m = %ld + [0, %'ld)\n",
            config.p, config.d, config.mstart, config.minc);
    }

    if (config.mskip > 0) {
        printf("\tskipping m < %'ld\n", config.mskip);
        assert(config.mskip >= config.mstart);
        assert(config.mskip < (config.mstart + config.minc));
    }

    setlocale(LC_NUMERIC, "C");

    // Determine compression
    {
        std::string fn = Args::gen_unknown_fn(config, ".txt");
        std::ifstream unknown_file(fn, std::ios::in);
        assert( unknown_file.is_open() ); // Can't open save_unknowns file
        assert( unknown_file.good() );    // Can't open save_unknowns file
        config.compression = Args::guess_compression(config, unknown_file);
    }

    prime_gap_test(config);
}


class GPUBatch {
    public:
        enum State { EMPTY, READY, RESULT_WRITTEN };
        State state = EMPTY;

        // current index;
        int i;

        // number to check if prime
        vector<mpz_t*> z;
        // XXX: This is an ugly hack because you can't create mpz_t vector easily
        mpz_t *z_array;

        // If z[i] should be tested
        vector<bool>  active;
        // Result from GPU
        vector<int>  result;

        // index into 'processing' (DataM)
        vector<int64_t> data_i;

        vector<int> unknown_i;

        GPUBatch(size_t n) {
            elements = n;

            z_array = (mpz_t *) malloc(n * sizeof(mpz_t));
            for (size_t i = 0; i < n; i++) {
                // MAYBE FIXES MY STALL ISSUE?
                mpz_init(z_array[i]);
                z.push_back(&z_array[i]);
            }

            active.resize(n, 0);
            result.resize(n, -1);

            data_i.resize(n, -1);
            unknown_i.resize(n, -1);
        }

        ~GPUBatch() {
            for (size_t i = 0; i < elements; i++) {
                mpz_clear(z_array[i]);
            }
        }

    private:
        size_t elements;
};

class DataM {
    public:
        /**
         * Elements in READY state can ONLY be modified by load_thread
         * Elements in RUNNING are either part of a GPU batch in overflowed queue
         */
        DataM() {};
        DataM(long m): m(m) {};

        enum State { READY, RUNNING, OVERFLOW_DONE };
        State state = READY;

        long m;
        mpz_t center;
        vector<int32_t> unknowns[2];

        bool p_found = false, n_found = false;
        int prev_p = 0, next_p = 0;

        // if this entry needs to be handled manually
        bool overflow = false;

        size_t p_tests = 0;
        size_t n_tests = 0;
};


/** Shared state between threads */
std::atomic<bool> is_running;

/**
 * Note: Uses a double batched system
 * C++ Thread is preparing batch_a (even more m)
 * While GPU runs batch_b
 */
vector<GPUBatch> batches = {{BATCH_GPU}, {BATCH_GPU}};

std::mutex overflow_mtx;
std::condition_variable overflow_cv;
vector<std::shared_ptr<DataM>> overflowed;

void run_gpu_thread(const struct Config config) {
    pthread_setname_np(pthread_self(), "RUN_GPU_THREAD");

    // XXX: params1024, params2048 with *runner1024, *runner2048 and only new one of them.
    typedef mr_params_t<THREADS_PER_INSTANCE, BITS, WINDOW_BITS> params;
    test_runner_t<params> runner(BATCH_GPU, ROUNDS);

    size_t processed_batches = 0;
    size_t no_batch_count_ms = 0;
    while (is_running) {
        bool no_batch = true;
        for (GPUBatch& batch : batches) {
            if (batch.state == GPUBatch::State::READY) {
                if (batch.i != BATCH_GPU) {
                    size_t test_active = 0;
                    for (size_t gpu_i = 0; gpu_i < BATCH_GPU; gpu_i++) {
                        test_active += batch.active[gpu_i];
                        if (!batch.active[gpu_i]) {
                            // This prevents the GPU from stalling if z was never initalized.
                            mpz_set_ui(*batch.z[gpu_i], 7);
                        }
                    }
                    printf("Partial batch %d/%ld actual: %lu\n", batch.i, BATCH_GPU, test_active);
                }
                // Run batch on GPU and wait for results to be set
                runner.run_test(batch.z, batch.result);
                batch.state = GPUBatch::State::RESULT_WRITTEN;
                no_batch = false;
                processed_batches++;
            }
        }
        if (no_batch) {
            // Waiting doesn't count till 1st batch is ready
            if (config.verbose >= 0 && processed_batches > 0) {
                no_batch_count_ms += 100;
                printf("Waiting on batch%ld => %.1f seconds\n",
                        no_batch_count_ms / 100, no_batch_count_ms / 1000.0);
            }
            usleep(250000); // 250ms
        }
    }

    if (config.verbose >= 1) {
        printf("Processed %'ld batches\n", processed_batches);
    }
}

void run_overflow_thread(const struct Config config) {
    mpz_t prime_test;
    mpz_init(prime_test);

    std::unique_lock<std::mutex> lock(overflow_mtx);

    while (true) {
        overflow_cv.wait(lock, []{ return overflowed.size() || !is_running; });
        if (!is_running) break;

        while (overflowed.size()) {
            DataM& interval = *overflowed.back(); overflowed.pop_back();
            lock.unlock();  // Allow main thread to add more things while we process
            assert (interval.overflow && interval.state == DataM::State::RUNNING);

            // NOTE: Overhead to doing this while GPU waits seems small (<1% of candidates)
            // But is actually A LOT because 40x slower. Becomes ~20-40% overhead quickly.

            if (interval.next_p == -1) {
                assert(interval.n_tests > 0);

                //cout << "gap_out_of_sieve_next m=" << interval.m << endl;
                mpz_add_ui(prime_test, interval.center, config.sieve_length);
                mpz_nextprime(prime_test, prime_test);
                mpz_sub(prime_test, prime_test, interval.center);

                interval.next_p = mpz_get_ui(prime_test);
                //cout << "gap_out_of_sieve_next m=" << interval.m << " -> " << interval.next_p << endl;
                interval.n_found = true;
                interval.overflow = 0;
            }
            if (interval.prev_p == -1) {
                // It took two years to mpz_prevprime into gmp.
                // I'm so proud and excited to get to use it here.

                assert(interval.p_tests == 0);
                //cout << "gap_out_of_sieve_prev m=" << interval.m << endl;
                mpz_prevprime(prime_test, interval.center);
                mpz_sub(prime_test, interval.center, prime_test);

                interval.prev_p = mpz_get_ui(prime_test);
                //cout << "gap_out_of_sieve_prev m=" << interval.m << " -> " << interval.prev_p << endl;
                interval.p_found = true;
                interval.overflow = 0;
            }

            // Mark interval as finished processing
            // NOTE: don't mark as READY or race_condition can happen in load
            interval.state = DataM::State::OVERFLOW_DONE;

            lock.lock(); // Lock so that overflow_cv / unlock waits correctly
        }
    }

    mpz_clear(prime_test);
}

void load_batch_thread(const struct Config config, const size_t QUEUE_SIZE) {
    // TODO ask C++ person if I need to worry about CPU doing cache invalidation with this setup
    // if batch is RESULT_WRITTEN | read result back to DataM processing | update to EMPTY
    // if batch is EMPTY          | load data from DataM processing      | update to READY, unlock GPU thread
    // if all batches EMPTY, wait(thread_sync)

    mpz_t K;
    double K_log;
    std::ifstream unknown_file;

    // Used for various stats
    StatsCounters stats(high_resolution_clock::now());

    std::unordered_map<int64_t, std::shared_ptr<DataM>> processing;

    const uint64_t P = config.p;
    const uint64_t D = config.d;
    const uint64_t M_start = config.mstart;
    const uint64_t M_inc = config.minc;

    const float min_merit = config.min_merit;

    // Print Header info & Open unknown_fn
    {

        // ----- Merit / Sieve stats
        K_log = prob_prime_and_stats(config, K);
        {
            float m_log = log(M_start);
            if (config.verbose >= 1) {
                printf("Min Gap ~= %d (for merit > %.1f)\n",
                    (int) (min_merit * (K_log + m_log)), min_merit);
            }
        }

        // ----- Open unknown input file
        {
            std::string fn = Args::gen_unknown_fn(config, ".txt");
            if (config.verbose >= 1) {
                printf("\nReading unknowns from '%s'\n", fn.c_str());
            }
            unknown_file.open(fn, std::ios::in);
            assert( unknown_file.is_open() ); // Can't open save_unknowns file
            assert( unknown_file.good() );    // Can't open save_unknowns file
        }

        uint64_t first_mi = 0;
        for (; first_mi > 0 && gcd(M_start + first_mi, D) > 1; first_mi++);
        assert(first_mi < M_inc);

        uint64_t last_mi = M_inc - 1;
        for (; last_mi > 0 && gcd(M_start + last_mi, D) > 1; last_mi--);
        assert(last_mi > 0 && last_mi < M_inc);

        // ----- Main sieve loop.
        if (config.verbose >= 1) {
            uint64_t valid_ms = count_num_m(M_start, M_inc, D);
            assert(valid_ms > 0 && valid_ms <= M_inc);

            printf("\n%ld tests M_start(%ld) + mi(%ld to %ld)\n\n",
                valid_ms, M_start, first_mi, last_mi);
        }
    }

    // For compressed lines
    BitArrayHelper helper(config, K);

    // Main loop
    uint64_t mi = 0;
    while (mi < M_inc || !processing.empty()) {
        usleep(500); // 0.5ms
        for (GPUBatch& batch : batches) {
            // If batch is ready to have new data loaded
            if (batch.state == GPUBatch::State::EMPTY) {
                // Add new DataM if free space
                for (; processing.size() < QUEUE_SIZE && mi < M_inc; mi++) {
                    uint64_t m = M_start + mi;
                    if (gcd(m, D) > 1) continue;

                    std::string line;
                    // Loop can be pragma omp parallel if this is placed in critical section
                    std::getline(unknown_file, line);

                    std::istringstream iss_line(line);

                    // Can skip if m < M_RESUME without parsing line here
                    if (m < config.mskip) continue;

                    auto test = std::make_shared<DataM>(m);

                    uint64_t m_parsed = parse_unknown_line(
                        config, helper, m, iss_line, test->unknowns[0], test->unknowns[1]);
                    assert(m_parsed == (uint64_t) m);

                    mpz_init(test->center);
                    mpz_mul_ui(test->center, K, test->m);

                    processing[test->m] = test;
                }

                // Grap some entries from each item in M
                {
                    batch.i = 0;
                    // Turn off all entries in batch
                    std::fill_n(batch.active.begin(), BATCH_GPU, false);
                    // Mark all results as invalid
                    std::fill_n(batch.result.begin(), BATCH_GPU, -1);

                    for (auto& pair : processing) {
                        auto& interval = *pair.second;
                        if (interval.state != DataM::State::READY or interval.overflow) {
                            // Already part of some other batch
                            continue;
                        }

                        for (size_t j = 0; j < SEQUENTIAL_IN_BATCH; j++) {
                            assert(! (interval.p_found && interval.n_found) );

                            int gpu_i = batch.i;  // [GPU] batch index
                            batch.data_i[gpu_i] = interval.m;  // [Data] index for GPU Batch

                            // One sided only runs positive side.
                            assert(!interval.n_found);
                            if (interval.n_tests < interval.unknowns[1].size()) {
                                mpz_add_ui(*batch.z[gpu_i], interval.center, interval.unknowns[1][interval.n_tests]);
                                batch.unknown_i[gpu_i] = interval.n_tests++;
                            } else {
                                // Haven't found next prime, but run out of unknowns to test
                                interval.next_p = -1;
                                interval.overflow = 1; // Indicates next side has overflowed unknowns
                                break;
                            }

                            //gmp_printf("batch[%d] = %d,%d = %d | %Zd\n", gpu_i, i, j, interval.m, *batch.z[gpu_i]);
                            interval.state = DataM::State::RUNNING;
                            batch.active[gpu_i] = true;
                            batch.i++;
                            if (batch.i == BATCH_GPU) break;
                        }
                        if (batch.i == BATCH_GPU) break;
                    }

                    // Every batch should be full unless we are almost done
                    // technically if many overflowed results this could not be true.
                    assert( (mi >= M_inc) || (batch.i == BATCH_GPU) );
                }

                // Mark batch as ready for GPU processing
                batch.state = GPUBatch::State::READY;
            }

            // If PRP result has been written to all entries by GPU
            if (batch.state == GPUBatch::State::RESULT_WRITTEN) {
                // Read results, mark any found primes, and possible finalize m-interval
                {
                    for (size_t i = 0; i < BATCH_GPU; i++) {
                        if (!batch.active[i]) {
                            continue;
                        }
                        // Verify GPU really did write the result
                        assert (batch.result[i] == 0 || batch.result[i] == 1);

                        DataM &interval = *processing.at(batch.data_i[i]);
                        // Mark interval as being ready again
                        interval.state = DataM::State::READY;

                        if (batch.result[i]) {
                            // Found prime in last partial batch of unknowns, no longer overflowed
                            interval.overflow = 0;

                            int offset_i = batch.unknown_i[i];
                            if (interval.n_found) {
                                /*
                                cout << "Found two next primes for m=" << interval.m << endl;
                                cout << "\t" << interval.next_p << " vs "
                                     << interval.unknowns[1][offset_i] << "(" << offset_i << ")" << endl;
                                */
                                continue;
                            }

                            // next_prime found (and done)
                            assert(interval.n_tests > 0 );
                            interval.n_found = true;
                            interval.next_p = interval.unknowns[1][offset_i];
                        }
                    }
                }

                // Finalize any finished (or overflowed) results from processing
                {
                    // Push Out-Of-Sieve gaps to overflow queue and notify that thread
                    {
                        bool pushed_to_overflow = false;
                        for (auto& pair : processing) {
                            auto& interval = *pair.second;
                            if (interval.overflow && interval.state == DataM::State::READY) {
                                if (interval.next_p == -1) {
                                    assert(interval.n_tests > 0);
                                    stats.s_gap_out_of_sieve_next += 1;
                                }

                                if (interval.prev_p == -1) {
                                    assert(interval.n_found);
                                    stats.s_gap_out_of_sieve_prev += 1;
                                }

                                // Push to overflow and wake up that thread
                                interval.state = DataM::State::RUNNING;
                                {
                                    std::unique_lock<std::mutex> lock(overflow_mtx);
                                    // TODO I THINK THERE'S BAD WRITES IF OVERFLOWED CHANGES POINTS AND STUFF
                                    // MAYBE I CAN MAKE SHARED POINTERS OR SOMETHING
                                    // OR I CAN PUSH TO A DIFFERENT QUEUE TBD NOT SURE
                                    overflowed.push_back(pair.second);
                                    pushed_to_overflow = true;
                                }
                            }
                        }
                        // TODO print warning if overflowed.size() is very large
                        if (pushed_to_overflow) {
                            overflow_cv.notify_one();
                        }
                    }

                    {
                        // Update any items finished in overflow as ready to be loaded into batches again
                        for (auto& pair : processing) {
                            auto& interval = *pair.second;
                            if (interval.state == DataM::State::OVERFLOW_DONE) {
                                interval.state = DataM::State::READY;
                            }
                        }
                    }

                    // Ugly code that allows for remove during iteration
                    auto it = processing.begin();
                    while (it != processing.end()) {
                        auto& interval = *it->second;

                        int prev_p = interval.prev_p;
                        int next_p = interval.next_p;

                        // Potentially do Side-Skip if next_p is not very large.
                        // Only consider if next_p just found.
                        if (interval.n_found && interval.prev_p == 0 && !interval.p_found) {
                            // TODO improve this with constant and logging
                            float next_merit = next_p / (K_log + log(interval.m));
                            /**
                             * TODO better math
                             * With Y = 24
                             * 50% of gaps with merit > 24 merit have prev > 12 merit
                             *      only test 1/2^(12-3) = 1/512 gaps
                             * 75% of gaps with merit > 24 merit have prev > 6 merit
                             *      test 1/2^(6-3) = 1/8 gaps
                             */
                            float MIN_MERIT_TO_CONTINUE = min_merit / 2 - 2;

                            if (next_merit < MIN_MERIT_TO_CONTINUE) {
                                stats.s_skips_after_one_side += 1;

                                bool is_last = (mi >= M_inc) && processing.size() == 1;
                                stats.process_results(config, interval.m, is_last,
                                    interval.unknowns[0].size(), interval.unknowns[1].size(),
                                    prev_p, next_p,
                                    interval.p_tests, interval.n_tests, next_merit);

                                mpz_clear(interval.center);
                                it = processing.erase(it);  // Erase this element
                                continue;
                            }

                            //cout << "Queued prev_p for check " << interval.m << endl;

                            // Mark this for overflow
                            // TODO not clear if it's bad that not pushed to overflow here.
                            interval.prev_p = -1;
                            interval.overflow = 1; // Indicates a side has overflowed
                            continue;
                        }

                        if (!interval.p_found || !interval.n_found) {
                            ++it;
                            continue;
                        }
                        assert( prev_p > 0 && next_p > 0 );

                        float merit = (next_p + prev_p) / (K_log + log(interval.m));
                        if (merit > min_merit)  {
                            // TODO: Record finished mi in log file / db.
                            printf("%-5d %.4f  %ld * %ld#/%ld -%d to +%d\n",
                                (next_p + prev_p), merit, interval.m, P, D, prev_p, next_p);
                        }

                        bool is_last = (mi >= M_inc) && processing.size() == 1;
                        stats.process_results(config, interval.m, is_last,
                            interval.unknowns[0].size(), interval.unknowns[1].size(),
                            prev_p, next_p,
                            interval.p_tests, interval.n_tests, merit);

                        mpz_clear(interval.center);
                        it = processing.erase(it);  // Erase this element
                    }
                }

                // Result batch to EMPTY
                batch.state = GPUBatch::State::EMPTY;
            }
        }
    }

    // ----- cleanup
    {
        mpz_clear(K);
    }
}


void prime_gap_test(struct Config config) {
    // Setup test runner
    printf("BITS=%d\tWINDOW_BITS=%d\n", BITS, WINDOW_BITS);
    printf("PRP/BATCH=%ld\tM/BATCH=%ld\n",
            BATCH_GPU, BATCH_GPU/SEQUENTIAL_IN_BATCH);
    printf("THREADS/PRP=%d\n", THREADS_PER_INSTANCE);

    assert( BATCH_GPU == 1024 || BATCH_GPU == 2048 || BATCH_GPU == 4096 ||
            BATCH_GPU == 8192 || BATCH_GPU ==16384 || BATCH_GPU ==32768 );
    assert( SEQUENTIAL_IN_BATCH == 1 || SEQUENTIAL_IN_BATCH == 2 || SEQUENTIAL_IN_BATCH == 4 );

    {
        mpz_t K;
        init_K(config, K);
        size_t N_bits = mpz_sizeinbase(K, 2) + log2(config.mstart + config.minc);
        mpz_clear(K);

        // P# roughly 349, 709, 1063, 1447
        for (size_t bits : {512, 1024, 1536, 2048, 3036, 4096}) {
            if (N_bits <= bits) {
                if (bits < BITS) {
                    printf("\nFASTER WITH `make gap_test_gpu BITS=%ld` (may require `make clean`)\n\n", bits);
                }
                break;
            }
        }
        assert( N_bits < BITS ); // See last debug line.
        assert( BITS <= (1 << (2 * WINDOW_BITS)) );
    }

    is_running = true;

    std::thread load_thread(load_batch_thread, config, BATCHED_M);
    std::thread gpu_thread(run_gpu_thread, config);
    std::thread overflow_thread(run_overflow_thread, config);

    load_thread.join();

    is_running = false;
    overflow_cv.notify_one();  // wake up overflow thread

    gpu_thread.join();
    overflow_thread.join();
}

